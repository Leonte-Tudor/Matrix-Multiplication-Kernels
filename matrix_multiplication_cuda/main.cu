
#include <hip/hip_runtime.h>
#include <iostream>
#include <time.h>

#define BLOCK_DIM 8
const unsigned int TILE_DIM = 8;

template <typename T>
__global__ void mm_tiled_kernel(T* A_d, T* B_d, T* C_d, const unsigned int m, const unsigned int n, const unsigned int p){

    __shared__ T A_s[TILE_DIM][TILE_DIM];
    __shared__ T B_s[TILE_DIM][TILE_DIM];

    unsigned int row = blockIdx.y * blockDim.y + threadIdx.y;
    unsigned int col = blockIdx.x * blockDim.x + threadIdx.x;

    float sum = 0.0f;

    for(unsigned int tile = 0; tile < m/TILE_DIM; ++tile){

        A_s[threadIdx.y][threadIdx.x] = A_d[row * p + tile * TILE_DIM + threadIdx.x];
        B_s[threadIdx.y][threadIdx.x] = B_d[(tile * TILE_DIM + threadIdx.y) * n + col];
        __syncthreads();

        for (unsigned int i = 0; i < TILE_DIM; ++i)
            sum += A_s[threadIdx.y][i] * B_s[i][threadIdx.x];
        __syncthreads();

    }

    C_d[row * n + col] = sum;

}

int main() {

    const unsigned int m = 2048, n = 1024, p = 512; // select matrix sizes (should be divisible by TILE_DIM)
    unsigned int i, c;
    auto* A = new float [m * p];
    auto* B = new float [p * n];
    auto* C = new float [m * n];

    for(i = 0; i < m * p; ++i)
        //if(i/p == i%p) // this condition will produce the identity matrix (multiplied by some scalar) when A is square
        A[i] = 3;

    for(i = 0; i < p * n; ++i)
        B[i] = 2;

    float *A_d, *B_d , *C_d;
    hipMalloc(&A_d, m * p * sizeof(float));
    hipMalloc(&B_d, p * n * sizeof(float));
    hipMalloc(&C_d, m * n * sizeof(float));

    hipMemcpy(A_d, A, m * p * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, p * n * sizeof(float), hipMemcpyHostToDevice);

    const dim3 ThreadsPerBlock (BLOCK_DIM, BLOCK_DIM, 1);
    const dim3 BlocksPerGrid (n/BLOCK_DIM, m/BLOCK_DIM, 1);

    clock_t start = clock();

    mm_tiled_kernel<<<BlocksPerGrid,ThreadsPerBlock>>>(A_d, B_d, C_d, m, n, p);
    hipDeviceSynchronize();

    clock_t stop = clock();
    double elapsed = (double) (stop - start) / CLOCKS_PER_SEC;
    printf("\nTime elapsed: %.5f\n", elapsed);

    hipMemcpy(C, C_d, m * n * sizeof(float), hipMemcpyDeviceToHost);

    // printing useful for smaller matrices
    /*c = 1;
    for(i = 0 ; i < m * p; ++i) {
        printf("%f ", A[i]);
        if(c%p == 0)
            printf("\n");
        ++c;
    }
    printf("\n");

    c = 1;
    for(i = 0 ; i < p * n; ++i) {
        printf("%f ", B[i]);
        if(c%n == 0)
            printf("\n");
        ++c;
    }
    printf("\n");

    c = 1;
    for(i = 0 ; i < m * n; ++i) {
        printf("%f ", C[i]);
        if(c%n == 0)
            printf("\n");
        ++c;
    }*/

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    delete[] A;
    delete[] B;
    delete[] C;


}
